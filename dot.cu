#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREAD 128

__global__ void dot(int N,float *x,float*y,float *ans);

int main(void){
    /*for CPU*/
    int i;
    int size = 1024;
    int block = (size + THREAD -1);//number of block

    float *x,*y,*ans;//(x,y)
    float z;
    
    hipHostMalloc((void **)&x,sizeof(float)*size, hipHostMallocDefault);
    hipHostMalloc((void **)&y,sizeof(float)*size, hipHostMallocDefault);
    hipHostMalloc((void **)&ans,sizeof(float)*block, hipHostMallocDefault);
    
    
    /*fo GPU*/
    float *d_x,*d_y,*d_ans;
    hipMalloc((void **)&d_x,sizeof(float)*size);
    hipMalloc((void **)&d_y,sizeof(float)*size);
    hipMalloc((void **)&d_ans,sizeof(float)*block);
    for(i=0;i<size;i++){
        x[i]=1.0;
        y[i]=1.0;
        }    
    
    /*Memory copy Host to Device*/
    
    hipMemcpy(d_x,x,sizeof(float)*size,hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,sizeof(float)*size,hipMemcpyHostToDevice);
    
    dot<<<block,THREAD>>>(size,d_x,d_y,d_ans);

    /*Memory copy Device to Host*/

    hipMemcpy(ans,d_ans,sizeof(float)*block,hipMemcpyDeviceToHost);
    z = 0.0;
    for(i=0;i<block;i++)
        z+=ans[i];
    //show answer
    printf("%f\n",z);
    
    /*CPU Memory free*/
    hipFree(x);
    hipFree(y);
    hipFree(d_ans);


    /*GPU Memory free*/
   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_ans);
    return 0;  
  }


__global__ void dot(int N,float *x,float *y,float *ans){
    int i,j;
    __shared__ float tmp[THREAD];
    
    tmp[threadIdx.x]=0;
    j = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(j<N){
     tmp[threadIdx.x] += x[j] * y[j];
    }
    else {
     tmp[threadIdx.x] =0.0;    
    }
    for(i = THREAD/2;i>31;i=i/2){
        if(threadIdx.x<i){
            tmp[threadIdx.x] += tmp[threadIdx.x+i];
    __syncthreads();
    }
}
    if(threadIdx.x<16){
        tmp[threadIdx.x] += tmp[threadIdx.x + 16];
         __syncthreads();
        tmp[threadIdx.x] += tmp[threadIdx.x + 8];
         __syncthreads();
        tmp[threadIdx.x] += tmp[threadIdx.x + 4];
         __syncthreads();
        tmp[threadIdx.x] += tmp[threadIdx.x + 2];
         __syncthreads();
        tmp[threadIdx.x] += tmp[threadIdx.x + 1];
         __syncthreads();


        }
    if(threadIdx.x == 0){
        ans[blockIdx.x] = tmp[0];    
        }
    }

