#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREAD 128

//texture<int2, 1, cudaReadModeElementType> yoshi;
	
__global__ void gemv(int n, double *adim, double *b, double *d_ans);

void cgemv(int n, double *adim, double *b, double *d_ans);

double gettime()
{
struct timeval tv;
gettimeofday(&tv, NULL);
return tv.tv_sec + (double)tv.tv_usec*1.0e-6;
}

int main(int argc, char **argv)
{
/* for CPU */
int i, j;
double *bdim, *c, *ans;
//double start, stop;
//double cpu_time, gpu_time;
int n = 1024;

bdim = (double *)malloc(sizeof(double) *n*n);
c = (double *)malloc(sizeof(double) *n);
ans = (double *)malloc(sizeof(double) *n);

/* for GPU */
double *d_bdim, *d_c, *d_ans;
hipMalloc((void **)&d_bdim, sizeof(double)*n*n);
hipMalloc((void **)&d_c, sizeof(double)*n);
hipMalloc((void **)&d_ans, sizeof(double)*n);

for(i = 0; i < n; i++)
{
c[i] = 1.0;
for(j = 0; j < n; j++)
bdim[i*n+j] = 1.0;
}

/*start = gettime();
cgemv(n, bdim, c, ans);
stop = gettime();

cpu_time=stop - start;
*/
hipMemcpy(d_bdim, bdim, sizeof(double)*n*n, hipMemcpyHostToDevice);
hipMemcpy(d_c, c, sizeof(double)*n, hipMemcpyHostToDevice);

//cudaBindTexture(0, yoshi, d_c, sizeof(double)*n);

//start = gettime();
gemv<<<n, THREAD>>>(n, d_bdim, d_c, d_ans);
//stop = gettime();

//gpu_time=stop - start;

hipMemcpy(ans, d_ans, sizeof(double)*n, hipMemcpyDeviceToHost);

//printf("cpu_time : %.6f[sec]\n",cpu_time);
//printf("gpu_time : %.6f[sec]\n",gpu_time);
//printf("%f x\n", cpu_time / gpu_time);


for(i = 0; i < n; i++)
printf("%f\n", ans[i]);


free(bdim);
free(c);
free(ans);
hipFree(d_bdim);
hipFree(d_c);
hipFree(d_ans);

return 0;
} 

__global__ void gemv(int n, double *adim, double *b, double *d_ans)
{
int i;
int div = n/THREAD;
//int2 fjt;
__shared__ double tmp[THREAD];

tmp[threadIdx.x] = 0.0;

for(i = 0; i < div; i++)
{
  /*fjt = tex1Dfetch(yoshi, i*THREAD+threadIdx.x); */
tmp[threadIdx.x] += adim[blockIdx.x*n+i*THREAD+threadIdx.x] * b[i * THREAD + threadIdx.x];
}
//fjt = tex1Dfetch(yoshi,div*THREAD+threadIdx.x);
if(threadIdx.x < n%THREAD)
tmp[threadIdx.x] += adim[blockIdx.x*n+THREAD*div+threadIdx.x] * b[THREAD * div + threadIdx.x];

__syncthreads();

for(i = THREAD / 2; i > 31; i = i / 2)
{
if(threadIdx.x < i)
tmp[threadIdx.x] += tmp[threadIdx.x + i];
__syncthreads();
}

if(threadIdx.x < 16)
{
tmp[threadIdx.x] += tmp[threadIdx.x + 16];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 8];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 4];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 2];
__syncthreads();
tmp[threadIdx.x] += tmp[threadIdx.x + 1];
__syncthreads();
}


if(threadIdx.x == 0)
d_ans[blockIdx.x] = tmp[0];

}

void cgemv(int n, double *adim, double *b, double *d_ans)
{
int i, j;

for(i = 0; i < n; i++)
for(j = 0; j < n; j++)
d_ans[i] = adim[i*n+j] * b[i];

}
